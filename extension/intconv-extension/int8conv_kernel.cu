#include <torch/extension.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <ATen/cudnn/Handle.h>

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << hipdnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

#define CUDA_CALL(f) { \
  hipError_t err = (f); \
  if (err != hipSuccess) { \
    std::cout \
        << "    Error occurred: " << err << std::endl; \
    std::exit(1); \
  } \
}

void find_algo(
        torch::Tensor& input, 
        torch::Tensor& weight,
        int32_t stride,
        int32_t padding,
        int32_t dilation,
        int32_t float_flag){

    /* only support n_in and c_in multiply of 4 */

    hipdnnHandle_t cudnnHandle = at::native::getCudnnHandle();

    hipdnnDataType_t input_type;
    hipdnnDataType_t output_type;
    hipdnnDataType_t conv_type;
    hipdnnTensorFormat_t input_format;
    hipdnnTensorFormat_t output_format;

    if (float_flag == 0){
        conv_type = HIPDNN_DATA_INT32;

        input_type = HIPDNN_DATA_INT8;
        // input_format = HIPDNN_TENSOR_NCHW;
        input_format = HIPDNN_TENSOR_NHWC;
        // input_type = HIPDNN_DATA_INT8x4;
        // input_format = HIPDNN_TENSOR_NCHW_VECT_C;

        //output_type = HIPDNN_DATA_INT8x4;
        //output_format = HIPDNN_TENSOR_NCHW_VECT_C;

        // output_type = HIPDNN_DATA_INT32;
        output_type = HIPDNN_DATA_FLOAT;
        output_format = HIPDNN_TENSOR_NHWC;
    }
    else{
        conv_type = HIPDNN_DATA_FLOAT;

        input_type = HIPDNN_DATA_FLOAT;
        input_format = HIPDNN_TENSOR_NHWC;

        output_type = HIPDNN_DATA_FLOAT;
        output_format = HIPDNN_TENSOR_NHWC;
    }

    int32_t n_in = input.size(0);
    int32_t h_in = input.size(1);
    int32_t w_in = input.size(2);
    int32_t c_in = input.size(3);
    hipdnnTensorDescriptor_t xDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&xDesc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(xDesc, 
                input_format, 
                input_type, 
                n_in, c_in, h_in, w_in));
    //std::cout<<n_in<<' '<<h_in<<' '<<w_in<<' '<<c_in<<' '<<std::endl;

    int32_t n_weight= weight.size(0);
    int32_t h_weight = weight.size(1);
    int32_t w_weight = weight.size(2);
    int32_t c_weight = weight.size(3);
    hipdnnFilterDescriptor_t wDesc;
    checkCUDNN(hipdnnCreateFilterDescriptor(&wDesc));
    checkCUDNN(hipdnnSetFilter4dDescriptor(wDesc, 
                input_type, 
                input_format, 
                n_weight, c_weight, h_weight, w_weight));

    //std::cout<<n_weight<<' '<<h_weight<<' '<<w_weight<<' '<<c_weight<<' '<<std::endl;

    hipdnnConvolutionDescriptor_t convDesc;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc, padding, padding, stride, stride, dilation, dilation, 
                HIPDNN_CROSS_CORRELATION,
                conv_type));

    //std::cout<<"create conv descriptor"<<std::endl;

    int32_t n_out;
    int32_t h_out;
    int32_t w_out;
    int32_t c_out;
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc, xDesc, wDesc, &n_out, &c_out, &h_out, &w_out));
    //std::cout<<n_out<<' '<<h_out<<' '<<w_out<<' '<<c_out<<' '<<std::endl;

    hipdnnTensorDescriptor_t yDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&yDesc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(yDesc, 
                output_format, 
                output_type, 
                n_out, c_out, h_out, w_out));

    //std::cout<<"create y tensor"<<std::endl;
    //auto y = torch::empty({n_out, h_out, w_out, c_out}, torch::dtype(torch::kFloat32).device(torch::kCUDA, 0));

    // my code
    hipdnnConvolutionFwdAlgoPerf_t algo;
    checkCUDNN(cudnnGetConvolutionForwardAlgorithm_v7(
        cudnnHandle,
        xDesc,
        wDesc,
        convDesc,
        yDesc,
        HIPDNN_CONVOLUTION_FWD_ALGO_GEMM,
        0,
        &algo
    ));
    std::cout<<"Convolution algorithm: "<<algo.algo<<std::endl;
    std::cout<<"==============="<<std::endl;


    // hipdnnConvolutionFwdAlgoPerf_t perfResults[3];
    // int32_t algo_cnt;
    // checkCUDNN(hipdnnFindConvolutionForwardAlgorithm(
    //             cudnnHandle, 
    //             xDesc, 
    //             wDesc, 
    //             convDesc, 
    //             yDesc, 
    //             3, 
    //             &algo_cnt, 
    //             perfResults));

    // std::cout<<"float flag: "<<float_flag<<std::endl;
    // std::cout<<"conv algorithm count: "<<algo_cnt<<std::endl;
    // for (int i=0; i<algo_cnt;i++){
    //     std::cout<<"algo: "<<perfResults[i].algo<<std::endl;
    //     std::cout<<"time: "<<perfResults[i].time<<std::endl;
    //     std::cout<<"memory: "<<perfResults[i].memory<<std::endl;
    //     std::cout<<"mathType: "<<perfResults[i].mathType<<std::endl;
    //     // std::cout<<"len perfResults: "<<perfResults[i].size()<<std::endl;
    //     sd::cout<<"==============="<<std::etndl;
    // }

    //hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
    //hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
    // this is algo 0
    //std::cout<<"algo: "<<HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM<<std::endl;
    // this is algo 1
    //std::cout<<"algo: "<<HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM<<std::endl;
    // according to find algo function, should use algo 1
}

torch::Tensor tensor_core_int8_conv(
        torch::Tensor& input, 
        torch::Tensor& weight,
        int32_t stride,
        int32_t padding,
        int32_t dilation){

    /* only support n_in and c_in multiply of 4 */

    hipdnnHandle_t cudnnHandle = at::native::getCudnnHandle();
    /* Tensor Descriptor 할당 */
    hipdnnTensorDescriptor_t xDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&xDesc));
    int32_t n_in = input.size(0);
    int32_t h_in = input.size(1);
    int32_t w_in = input.size(2);
    int32_t c_in = input.size(3);
    checkCUDNN(hipdnnSetTensor4dDescriptor(xDesc, 
                HIPDNN_TENSOR_NHWC, 
                HIPDNN_DATA_INT8, 
                n_in, c_in, h_in, w_in));


    int32_t n_weight= weight.size(0);
    int32_t h_weight = weight.size(1);
    int32_t w_weight = weight.size(2);
    int32_t c_weight = weight.size(3);
    
    /* filter Descriptor 할당 */
    hipdnnFilterDescriptor_t wDesc;
    checkCUDNN(hipdnnCreateFilterDescriptor(&wDesc));
    checkCUDNN(hipdnnSetFilter4dDescriptor(wDesc, 
                HIPDNN_DATA_INT8, 
                HIPDNN_TENSOR_NHWC, 
                n_weight, c_weight, h_weight, w_weight));

    /* Convolution setting 할당*/
    hipdnnConvolutionDescriptor_t convDesc;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc, padding, padding, stride, stride, dilation, dilation, 
                HIPDNN_CROSS_CORRELATION,
                HIPDNN_DATA_INT32));
    
    // if (c_in % 4 !=0){
    //     checkCUDNN(hipdnnSetConvolutionMathType(convDesc, CUDNN_FMA_MATH));
    // }

    int32_t n_out;
    int32_t h_out;
    int32_t w_out;
    int32_t c_out;
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc, xDesc, wDesc, &n_out, &c_out, &h_out, &w_out));

    /* 출력 구조 할당 */
    hipdnnTensorDescriptor_t yDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&yDesc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(yDesc, 
                HIPDNN_TENSOR_NHWC, 
                HIPDNN_DATA_INT32, 
                n_out, c_out, h_out, w_out));

    //std::cout<<"create y tensor"<<std::endl;
    auto y = torch::empty({n_out, h_out, w_out, c_out}, torch::dtype(torch::kInt32).device(torch::kCUDA, 0));

    // hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
    hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;

    float alpha = 1.0;
    //float alpha = 1.0;
    float beta = 0.0;

    //size_t ws_size = 355968;
    size_t ws_size;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,xDesc,wDesc,convDesc,yDesc,algo,&ws_size));
    auto workspace = torch::empty({static_cast<int64_t>(ws_size)}, torch::dtype(torch::kInt32).device(torch::kCUDA, 0));

    checkCUDNN(hipdnnConvolutionForward(cudnnHandle,
                &alpha,xDesc,input.data<int8_t>(),
                wDesc,weight.data<int8_t>(),
                convDesc,
                algo,
                workspace.data<int32_t>(),
                ws_size,
                &beta,yDesc,
                y.data<int32_t>()));

     checkCUDNN(hipdnnDestroyTensorDescriptor(yDesc));
     checkCUDNN(hipdnnDestroyConvolutionDescriptor(convDesc));
     checkCUDNN(hipdnnDestroyFilterDescriptor(wDesc));
     checkCUDNN(hipdnnDestroyTensorDescriptor(xDesc));

     return y;
}

torch::Tensor tensor_core_float_conv(
        torch::Tensor& input, 
        torch::Tensor& weight,
        int32_t stride,
        int32_t padding,
        int32_t dilation){

    /* only support n_in and c_in multiply of 4 */

    hipdnnHandle_t cudnnHandle = at::native::getCudnnHandle();
    /* Tensor Descriptor 할당 */
    hipdnnTensorDescriptor_t xDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&xDesc));
    int32_t n_in = input.size(0);
    int32_t h_in = input.size(1);
    int32_t w_in = input.size(2);
    int32_t c_in = input.size(3);
    checkCUDNN(hipdnnSetTensor4dDescriptor(xDesc, 
                HIPDNN_TENSOR_NHWC, 
                HIPDNN_DATA_FLOAT, 
                n_in, c_in, h_in, w_in));


    int32_t n_weight= weight.size(0);
    int32_t h_weight = weight.size(1);
    int32_t w_weight = weight.size(2);
    int32_t c_weight = weight.size(3);
    
    /* filter Descriptor 할당 */
    hipdnnFilterDescriptor_t wDesc;
    checkCUDNN(hipdnnCreateFilterDescriptor(&wDesc));
    checkCUDNN(hipdnnSetFilter4dDescriptor(wDesc, 
                HIPDNN_DATA_FLOAT, 
                HIPDNN_TENSOR_NCHW, 
                n_weight, c_weight, h_weight, w_weight));

    /* Convolution setting 할당*/
    hipdnnConvolutionDescriptor_t convDesc;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc, padding, padding, stride, stride, dilation, dilation, 
                HIPDNN_CROSS_CORRELATION,
                HIPDNN_DATA_FLOAT));
    
    // if (c_in % 4 !=0){
    //     checkCUDNN(hipdnnSetConvolutionMathType(convDesc, CUDNN_FMA_MATH));
    // }

    int32_t n_out;
    int32_t h_out;
    int32_t w_out;
    int32_t c_out;
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc, xDesc, wDesc, &n_out, &c_out, &h_out, &w_out));

    /* 출력 구조 할당 */
    hipdnnTensorDescriptor_t yDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&yDesc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(yDesc, 
                HIPDNN_TENSOR_NHWC, 
                HIPDNN_DATA_FLOAT, 
                n_out, c_out, h_out, w_out));

    //std::cout<<"create y tensor"<<std::endl;
    auto y = torch::empty({n_out, h_out, w_out, c_out}, torch::dtype(torch::kFloat32).device(torch::kCUDA, 0));

    // hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
    hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;

    float alpha = 1.0;
    //float alpha = 1.0;
    float beta = 0.0;

    //size_t ws_size = 355968;
    size_t ws_size;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,xDesc,wDesc,convDesc,yDesc,algo,&ws_size));
    auto workspace = torch::empty({static_cast<int64_t>(ws_size)}, torch::dtype(torch::kFloat32).device(torch::kCUDA, 0));

    checkCUDNN(hipdnnConvolutionForward(cudnnHandle,
                &alpha,xDesc,input.data<float>(),
                wDesc,weight.data<float>(),
                convDesc,
                algo,
                workspace.data<float>(),
                ws_size,
                &beta,yDesc,
                y.data<float>()));

     checkCUDNN(hipdnnDestroyTensorDescriptor(yDesc));
     checkCUDNN(hipdnnDestroyConvolutionDescriptor(convDesc));
     checkCUDNN(hipdnnDestroyFilterDescriptor(wDesc));
     checkCUDNN(hipdnnDestroyTensorDescriptor(xDesc));

     return y;
}